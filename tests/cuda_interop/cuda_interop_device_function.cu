#include "hip/hip_runtime.h"
/*******************************************************************************
//
//  SYCL 2020 Conformance Test Suite
//
//  Copyright:  (c) 2017 by Codeplay Software LTD. All Rights Reserved.
//
*******************************************************************************/

#include "../common/common.h"

#ifdef SYCL_EXT_ONEAPI_BACKEND_CUDA
#include "../../util/test_base_cuda.h"

#endif

#define TEST_NAME cuda_interop_device_fucntion

namespace cuda_interop_device_fucntion__ {

using namespace sycl_cts;

template <typename T>
SYCL_EXTERNAL __device__ T func(T *i) {
  return *i + 1;
}

/** check device fucntion can be called with the backend interop type `T *`
converted from an `accessor`
  */
template <typename T>
void test_device_function_buffer(sycl::queue &queue,
                                 sycl_cts::util::logger &log,
                                 const std::string &typeName) {
  size_t constexpr size = 1;
  T data[size];
  data[0] = 0;
  bool result[size] = {false};
  {
    sycl::buffer<T> buff(data, sycl::range<1>(size));
    sycl::buffer<bool> result_buf(result, sycl::range<1>(size));

    queue.submit([&](sycl::handler &cgh) {
      auto acc = buff.template get_access<sycl::access::mode::read>(cgh);
      auto result_acc = result_buf.get_access<sycl::access::mode::write>(cgh);

      cgh.single_task([=]() {
        result_acc[0] =
            func<T>(sycl::get_native<sycl::backend::ext_oneapi_cuda>(acc)) == 1;
      });
    });
  }

  if (!result[0]) {
    log.note(
        "Test for CUDA device function interop with accessor failed for \"" +
        typeName + "\" type");
  }
  assert(result[0]);
}

/** check device fucntion can be called with the backend interop type `T *`
converted from an `local_accessor`
  */
template <typename T>
void test_device_function_local(sycl::queue &queue, sycl_cts::util::logger &log,
                                const std::string &typeName) {
  size_t constexpr size = 1;
  bool result[size] = {false};
  {
    sycl::buffer<bool> result_buf(result, sycl::range<1>(size));

    queue.submit([&](sycl::handler &cgh) {
      sycl::local_accessor<T> acc(size, cgh);
      auto result_acc = result_buf.get_access<sycl::access::mode::write>(cgh);

      cgh.single_task([=]() {
        acc[0] = 0;
        result_acc[0] =
            func<T>(sycl::get_native<sycl::backend::ext_oneapi_cuda>(acc)) == 1;
      });
    });
  }

  if (!result[0]) {
    log.note(
        "Test for CUDA device function interop with local_accessor failed for "
        "\"" +
        typeName + "\" type");
  }
  assert(result[0]);
}

/** tests the get_native() methods for CUDA inter-op
 */
class TEST_NAME :
#ifdef SYCL_EXT_ONEAPI_BACKEND_CUDA
    public sycl_cts::util::test_base_cuda
#else
    public util::test_base
#endif
{
 public:
  /** return information about this test
   */
  void get_info(test_base::info &out) const override {
    set_test_info(out, TOSTRING(TEST_NAME), TEST_FILE);
  }

  /** execute this test
   */
  void run(util::logger &log) override {
#ifdef SYCL_EXT_ONEAPI_BACKEND_CUDA
    {
      auto queue = util::get_cts_object::queue();
      if (queue.get_backend() != sycl::backend::ext_oneapi_cuda) {
        WARN(
            "CUDA interoperability part is not supported on non-CUDA "
            "backend types");
        return;
      }
      cts_selector ctsSelector;

      test_device_function_buffer<int>(queue, log, "int");
      test_device_function_local<int>(queue, log, "int");
    }
#else
    log.note("The test is skipped because CUDA back-end is not supported");
#endif  // SYCL_EXT_ONEAPI_BACKEND_CUDA
  }
};

// register this test with the test_collection
util::test_proxy<TEST_NAME> proxy;

}  // namespace cuda_interop_device_fucntion__
